#include "hip/hip_runtime.h"
#include <iostream>
using namespace std;
#include <hip/hip_runtime.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <time.h>
#include "util.hpp"


// ----------------------------------------------------------------------------
// The adopted von Neuman neighborhood
// Format: flow_index:cell_label:(row_index,col_index)
//
//   cell_label in [0,1,2,3,4]: label assigned to each cell in the neighborhood
//   flow_index in   [0,1,2,3]: outgoing flow indices in Sf from cell 0 to the others
//       (row_index,col_index): 2D relative indices of the cells
//
//               |0:1:(-1, 0)|
//   |1:2:( 0,-1)| :0:( 0, 0)|2:3:( 0, 1)|
//               |3:4:( 1, 0)|
//
//
int h_Xi[] = {0, -1,  0,  0,  1};// Xj: von Neuman neighborhood row coordinates (see below)
int h_Xj[] = {0,  0, -1,  1,  0};// Xj: von Neuman neighborhood col coordinates (see below)
__constant__ int Xi[5]; // Xj: von Neuman neighborhood row coordinates (see below)
__constant__ int Xj[5]; // Xj: von Neuman neighborhood col coordinates (see below)
// ----------------------------------------------------------------------------
// I/O parameters used to index argv[]
// ----------------------------------------------------------------------------
#define HEADER_PATH_ID 1
#define DEM_PATH_ID 2
#define SOURCE_PATH_ID 3
#define OUTPUT_PATH_ID 4
#define STEPS_ID 5
#define BLOCK_SIZE_D0_ID 6
#define BLOCK_SIZE_D1_ID 7
// ----------------------------------------------------------------------------
// Simulation parameters
// ----------------------------------------------------------------------------
#define P_R 0.5
#define P_EPSILON 0.001
#define ADJACENT_CELLS 4
#define STRLEN 256

// ----------------------------------------------------------------------------
// Read/Write access macros linearizing single/multy layer buffer 2D indices
// ----------------------------------------------------------------------------
#define SET(M, columns, i, j, value) ((M)[(((i) * (columns)) + (j))] = (value))
#define GET(M, columns, i, j) (M[(((i) * (columns)) + (j))])
#define BUF_SET(M, rows, columns, n, i, j, value) ( (M)[( ((n)*(rows)*(columns)) + ((i)*(columns)) + (j) )] = (value) )
#define BUF_GET(M, rows, columns, n, i, j) ( M[( ((n)*(rows)*(columns)) + ((i)*(columns)) + (j) )] )

// ----------------------------------------------------------------------------
// I/O functions
// ----------------------------------------------------------------------------
void readHeaderInfo(char* path, int &nrows, int &ncols, /*double &xllcorner, double &yllcorner, double &cellsize,*/ double &nodata)
{
  FILE* f;
  
  if ( (f = fopen(path,"r") ) == 0){
    printf("%s configuration header file not found\n", path);
    exit(0);
  }

  //Reading the header
  char str[STRLEN];
  fscanf(f,"%s",&str); fscanf(f,"%s",&str); ncols = atoi(str);      //ncols
  fscanf(f,"%s",&str); fscanf(f,"%s",&str); nrows = atoi(str);      //nrows
  fscanf(f,"%s",&str); fscanf(f,"%s",&str); //xllcorner = atof(str);  //xllcorner
  fscanf(f,"%s",&str); fscanf(f,"%s",&str); //yllcorner = atof(str);  //yllcorner
  fscanf(f,"%s",&str); fscanf(f,"%s",&str); //cellsize = atof(str);   //cellsize
  fscanf(f,"%s",&str); fscanf(f,"%s",&str); nodata = atof(str);     //NODATA_value 
}

bool loadGrid2D(double *M, int rows, int columns, char *path)
{
  FILE *f = fopen(path, "r");

  if (!f) {
    printf("%s grid file not found\n", path);
    exit(0);
  }

  char str[STRLEN];
  for (int i = 0; i < rows; i++)
    for (int j = 0; j < columns; j++)
    {
      fscanf(f, "%s", str);
      SET(M, columns, i, j, atof(str));
    }

  fclose(f);

  return true;
}

bool saveGrid2Dr(double *M, int rows, int columns, char *path)
{
  FILE *f;
  f = fopen(path, "w");

  if (!f)
    return false;

  char str[STRLEN];
  for (int i = 0; i < rows; i++)
  {
    for (int j = 0; j < columns; j++)
    {
      sprintf(str, "%f ", GET(M, columns, i, j));
      fprintf(f, "%s ", str);
    }
    fprintf(f, "\n");
  }

  fclose(f);

  return true;
}

bool saveBinaryGrid2Dr(double *M, int rows, int columns, const char *path)
{
  FILE *f = fopen(path, "w");

  if (!f)
    return false;

  fwrite(M, sizeof(double), rows*columns, f);

  fclose(f);

  return true;
}

double* addLayer2D(int rows, int columns)
{
  //double *tmp = (double *)malloc(sizeof(double) * rows * columns);
  double *tmp;
  hipMallocManaged(&tmp, sizeof(double) * rows * columns);  
  if (!tmp)
    return NULL;
  return tmp;
}

// ----------------------------------------------------------------------------
// init kernel, called once before the simulation loop
// ----------------------------------------------------------------------------
void sciddicaTSimulationInit(int i, int j, int r, int c, double* Sz, double* Sh)
{
  double z, h;
  h = GET(Sh, c, i, j);

  if (h > 0.0)
  {
    z = GET(Sz, c, i, j);
    SET(Sz, c, i, j, z - h);
  }
}

// ----------------------------------------------------------------------------
// computing kernels, aka elementary processes in the XCA terminology
// ----------------------------------------------------------------------------
__global__
void sciddicaTResetFlows(int i_start, int i_end, int j_start, int j_end, int r, int c, double nodata, double* __restrict__ Sf)
{
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < i_start or i >= i_end or j < j_start or j >= j_end)
    return;

  BUF_SET(Sf, r, c, 0, i, j, 0.0);
  BUF_SET(Sf, r, c, 1, i, j, 0.0);
  BUF_SET(Sf, r, c, 2, i, j, 0.0);
  BUF_SET(Sf, r, c, 3, i, j, 0.0);
}

__global__
void sciddicaTFlowsComputation(int i_start, int i_end, int j_start, int j_end, int r, int c, double nodata, /*int* Xi, int* Xj,*/ const double* __restrict__ Sz, const double* __restrict__ Sh, double* __restrict__ Sf, const double p_r, const double p_epsilon)
{
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < i_start or i >= i_end or j < j_start or j >= j_end)
    return;

  //int Xi[] = {0, -1,  0,  0,  1};// Xj: von Neuman neighborhood row coordinates (see below)
  //int Xj[] = {0,  0, -1,  1,  0};// Xj: von Neuman neighborhood col coordinates (see below)

  bool eliminated_cells[5] = {false, false, false, false, false};
  bool again;
  int cells_count;
  double average;
  double m;
  double u[5];
  int n;
  double z, h;

  m = GET(Sh, c, i, j) - p_epsilon;
  if (m <= 0.0)
    return;

  u[0] = GET(Sz, c, i, j) + p_epsilon;
  z = GET(Sz, c, i + Xi[1], j + Xj[1]);
  h = GET(Sh, c, i + Xi[1], j + Xj[1]);
  u[1] = z + h;                                         
  z = GET(Sz, c, i + Xi[2], j + Xj[2]);
  h = GET(Sh, c, i + Xi[2], j + Xj[2]);
  u[2] = z + h;                                         
  z = GET(Sz, c, i + Xi[3], j + Xj[3]);
  h = GET(Sh, c, i + Xi[3], j + Xj[3]);
  u[3] = z + h;                                         
  z = GET(Sz, c, i + Xi[4], j + Xj[4]);
  h = GET(Sh, c, i + Xi[4], j + Xj[4]);
  u[4] = z + h;

  do
  {
    again = false;
    average = m;
    cells_count = 0;

    for (n = 0; n < 5; n++)
      if (!eliminated_cells[n])
      {
        average += u[n];
        cells_count++;
      }

    if (cells_count != 0)
      average /= cells_count;

    for (n = 0; n < 5; n++)
      if ((average <= u[n]) && (!eliminated_cells[n]))
      {
        eliminated_cells[n] = true;
        again = true;
      }
  } while (again);

  if (!eliminated_cells[1]) BUF_SET(Sf, r, c, 0, i, j, (average - u[1]) * p_r);
  if (!eliminated_cells[2]) BUF_SET(Sf, r, c, 1, i, j, (average - u[2]) * p_r);
  if (!eliminated_cells[3]) BUF_SET(Sf, r, c, 2, i, j, (average - u[3]) * p_r);
  if (!eliminated_cells[4]) BUF_SET(Sf, r, c, 3, i, j, (average - u[4]) * p_r);
}

__global__
void sciddicaTWidthUpdate(int i_start, int i_end, int j_start, int j_end, int r, int c, double nodata, /*int* Xi, int* Xj,*/ const double* __restrict__ Sz, double* __restrict__ Sh, const double* __restrict__ Sf)
{
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int j = blockIdx.x * blockDim.x + threadIdx.x;

  //int Xi[] = {0, -1,  0,  0,  1};// Xj: von Neuman neighborhood row coordinates (see below)
  //int Xj[] = {0,  0, -1,  1,  0};// Xj: von Neuman neighborhood col coordinates (see below)

  if (i < i_start or i >= i_end or j < j_start or j >= j_end)
    return;

  double h_next;
  h_next = GET(Sh, c, i, j);
  h_next += BUF_GET(Sf, r, c, 3, i+Xi[1], j+Xj[1]) - BUF_GET(Sf, r, c, 0, i, j);
  h_next += BUF_GET(Sf, r, c, 2, i+Xi[2], j+Xj[2]) - BUF_GET(Sf, r, c, 1, i, j);
  h_next += BUF_GET(Sf, r, c, 1, i+Xi[3], j+Xj[3]) - BUF_GET(Sf, r, c, 2, i, j);
  h_next += BUF_GET(Sf, r, c, 0, i+Xi[4], j+Xj[4]) - BUF_GET(Sf, r, c, 3, i, j);

  SET(Sh, c, i, j, h_next);
}

// ----------------------------------------------------------------------------
// Function main()
// ----------------------------------------------------------------------------
int main(int argc, char **argv)
{
  int rows, cols;
  double nodata;
  readHeaderInfo(argv[HEADER_PATH_ID], rows, cols, nodata);

  int r = rows;                  // r: grid rows
  int c = cols;                  // c: grid columns
  int i_start = 1, i_end = r-1;  // [i_start,i_end[: kernels application range along the rows
  int j_start = 1, j_end = c-1;  // [i_start,i_end[: kernels application range along the rows
  double *Sz;                    // Sz: substate (grid) containing the cells' altitude a.s.l.
  double *Sh;                    // Sh: substate (grid) containing the cells' flow thickness
  double *Sf;                    // Sf: 4 substates containing the flows towards the 4 neighs
  double p_r = P_R;              // p_r: minimization algorithm outflows dumping factor
  double p_epsilon = P_EPSILON;  // p_epsilon: frictional parameter threshold
  int steps = atoi(argv[STEPS_ID]); //steps: simulation steps
  int block_size_d0 = atoi(argv[BLOCK_SIZE_D0_ID]);
  int block_size_d1 = atoi(argv[BLOCK_SIZE_D1_ID]);

  Sz = addLayer2D(r, c);                 // Allocates the Sz substate grid
  Sh = addLayer2D(r, c);                 // Allocates the Sh substate grid
  Sf = addLayer2D(ADJACENT_CELLS* r, c); // Allocates the Sf substates grid, 
                                         //   having one layer for each adjacent cell

  loadGrid2D(Sz, r, c, argv[DEM_PATH_ID]);   // Load Sz from file
  loadGrid2D(Sh, r, c, argv[SOURCE_PATH_ID]);// Load Sh from file

  // Apply the init kernel (elementary process) to the whole domain grid (cellular space)
#pragma omp parallel for
    for (int i = i_start; i < i_end; i++)
      for (int j = j_start; j < j_end; j++)
        sciddicaTSimulationInit(i, j, r, c, Sz, Sh);

  hipMemcpyToSymbol(HIP_SYMBOL(Xi), h_Xi, 5*sizeof(int)); // Copy Xi to DEVICE in CONSTANT memory
  hipMemcpyToSymbol(HIP_SYMBOL(Xj), h_Xj, 5*sizeof(int)); // Copy Xj to DEVICE in CONSTANT memory

  hipMemPrefetchAsync(Sz, sizeof(double)*r*c, 0 , NULL);
  hipMemPrefetchAsync(Sh, sizeof(double)*r*c, 0 , NULL);
  hipMemPrefetchAsync(Sf, sizeof(double)*r*c*ADJACENT_CELLS, 0 , NULL);
  hipDeviceSynchronize();

  util::Timer cl_timer;
  // simulation loop
  for (int s = 0; s < steps; ++s)
  {
    dim3 block_size_rf(block_size_d0, block_size_d1, 1);
    dim3 grid_size_rf(ceil(c/(float)block_size_rf.x), ceil(r/(float)block_size_rf.y), 1);
    sciddicaTResetFlows<<<grid_size_rf,block_size_rf>>>(i_start, i_end, j_start, j_end, r, c, nodata, Sf);

    dim3 block_size_fc(block_size_d0, block_size_d1, 1);
    dim3 grid_size_fc(ceil(c/(float)block_size_fc.x), ceil(r/(float)block_size_fc.y), 1);
    sciddicaTFlowsComputation<<<grid_size_fc,block_size_fc>>>(i_start, i_end, j_start, j_end, r, c, nodata, /*Xi, Xj,*/ Sz, Sh, Sf, p_r, p_epsilon);

    dim3 block_size_wu(block_size_d0, block_size_d1, 1);
    dim3 grid_size_wu(ceil(c/(float)block_size_wu.x), ceil(r/(float)block_size_wu.y), 1);
    sciddicaTWidthUpdate<<<grid_size_wu,block_size_wu>>>(i_start, i_end, j_start, j_end, r, c, nodata, /*Xi, Xj,*/ Sz, Sh, Sf);
  }
  hipDeviceSynchronize();
  double cl_time = static_cast<double>(cl_timer.getTimeMilliseconds()) / 1000.0;
  printf(" %2d; %2d; %7.3f\n", block_size_d0, block_size_d1, cl_time);

  saveGrid2Dr(Sh, r, c, argv[OUTPUT_PATH_ID]);// Save Sh to file
  const std::string binPath = std::string(argv[OUTPUT_PATH_ID]) + ".bin";
  saveBinaryGrid2Dr(Sh, r, c, binPath.c_str());// Save Sh to file in binary format

  //printf("Releasing memory...\n");
  // delete[] Sz;
  // delete[] Sh;
  // delete[] Sf;
  hipFree(Sz);
  hipFree(Sh);
  hipFree(Sf);

  return 0;
}
